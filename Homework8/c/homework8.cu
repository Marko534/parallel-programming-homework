#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 1024

float *readVectorFromFile(FILE *file, int size)
{
    if (!file)
    {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    float *vector = (float *)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++)
    {
        if (fscanf(file, "%f", &vector[i]) != 1)
        {
            fclose(file);
            fprintf(stderr, "Error reading matrix data\n");
            exit(EXIT_FAILURE);
        }
    }

    return vector;
}

__global__ void blelloch_scan(float *input, float *output, int n)
{
    extern __shared__ float temp[]; // shared memory

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory and perform reduction (up-sweep) phase
    if (idx < n)
    {
        temp[tid] = input[idx];

        for (int d = 1; d < blockDim.x; d *= 2)
        {
            int ai = tid - d;
            if (ai >= 0)
            {
                temp[tid] += temp[ai];
            }
            __syncthreads();
        }
    }

    // Clear the last element
    if (tid == 0 && idx == n - 1)
    {
        temp[n - 1] = 0;
    }

    // Perform post-reduction (down-sweep) phase
    for (int d = blockDim.x / 2; d > 0; d /= 2)
    {
        __syncthreads();
        int ai = tid - d;
        if (ai >= 0)
        {
            float t = temp[ai];
            temp[ai] = temp[tid];
            temp[tid] += t;
        }
    }
    __syncthreads();

    // Write the result to output
    if (idx < n)
    {
        output[idx] = temp[tid];
    }
}

int main(int argc, char **argv)
{
    const char *filename = "vectors.txt";
    int VECTOR_LENGTH;
    FILE *file = fopen(filename, "r");
    if (!file)
    {
        perror("Error opening file");
        exit(EXIT_FAILURE);
    }

    if (fscanf(file, "%d", &VECTOR_LENGTH) != 1)
    {
        fclose(file);
        fprintf(stderr, "Error reading VECTOR_LENGTH\n");
        exit(EXIT_FAILURE);
    }

    printf("Vector length: %d\n", VECTOR_LENGTH);

    const size_t num_blocks = (VECTOR_LENGTH / BLOCK_SIZE) + ((VECTOR_LENGTH % BLOCK_SIZE) ? 1 : 0);

    float *h_input = readVectorFromFile(file, VECTOR_LENGTH);
    float *h_output = (float *)malloc(VECTOR_LENGTH * sizeof(float));
    float *d_input, *d_output;

    hipMalloc(&d_input, VECTOR_LENGTH * sizeof(float));
    hipMalloc(&d_output, VECTOR_LENGTH * sizeof(float));

    hipMemcpy(d_input, h_input, VECTOR_LENGTH * sizeof(float), hipMemcpyHostToDevice);

    blelloch_scan<<<num_blocks, BLOCK_SIZE, 2 * BLOCK_SIZE * sizeof(float)>>>(d_input, d_output, VECTOR_LENGTH);

    hipMemcpy(h_output, d_output, VECTOR_LENGTH * sizeof(float), hipMemcpyDeviceToHost);

    printf("Original vector and its inclusive scan result:\n");
    for (int i = 0; i < VECTOR_LENGTH; i++)
    {
        printf("%f\t%f\n", h_input[i], h_output[i]);
    }
    // printf("\n");

    // Free the memory
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    fclose(file);
    return 0;
}
